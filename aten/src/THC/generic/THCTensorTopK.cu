
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorTopK.cu"
#else

THC_API void THCTensor_(topk)(THCState* state,
                               THCTensor *topK,
                               THCudaLongTensor *indices,
                               THCTensor *input_,
                               int64_t k, int dim, int dir, int sorted) {
  THAssert(topK != NULL && indices != NULL && input_ != NULL);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, topK, indices, input_));
  THArgCheck(THCTensor_(nDimension)(state, topK) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  int64_t dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 3, CUTORCH_DIM_WARNING);
  int numDims = THCTensor_(nDimension)(state, input_);
  THArgCheck(numDims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);

  THArgCheck(dim >= 0 && dim < numDims, 6, "dim not in range");

  int64_t sliceSize = THCTensor_(size)(state, input_, dim);
  THArgCheck(k >= 0 && k <= sliceSize, 5, "k not in range for dimension");

  THCTensor *input = THCTensor_(newContiguous)(state, input_);

  // Build the output size, which is the dim being selected set to
  // size k
  THLongStorage* topKSize = THCTensor_(newSizeOf)(state, input);
  THLongStorage_set(topKSize, dim, k);
  THCTensor_(resize)(state, topK, topKSize, NULL);
  THCudaLongTensor_resize(state, indices, topKSize, NULL);
  THLongStorage_free(topKSize);

#define RUN_K(INDEX_T, DIM, DIR)                                        \
  gatherTopK<real, INDEX_T, DIM, DIR>                                   \
    <<<grid, block, 0, THCState_getCurrentStream(state)>>>(             \
      inputInfo,                                                        \
      sliceSize,                                                        \
      k,                                                                \
      inputSlices,                                                      \
      /* The actual dimension that the k-selection is running in */     \
      /* may have changed from collapseDims() */                        \
      inputInfo.strides[collapseInputDim],                              \
      topKInfo,                                                         \
      topKSlices,                                                       \
      topKInfo.strides[collapseTopKDim],                                \
      indicesInfo,                                                      \
      indicesInfo.strides[collapseIndicesDim])

#define RUN_DIR(INDEX_T, DIM)                   \
  if (dir) {                                    \
    RUN_K(INDEX_T, DIM, true);                  \
  } else {                                      \
    RUN_K(INDEX_T, DIM, false);                 \
  }

#define RUN_DIM(INDEX_T)                        \
  if (allDims == 1) {                           \
    RUN_DIR(INDEX_T, 1);                        \
  } else if (allDims == 2) {                    \
    RUN_DIR(INDEX_T, 2);                        \
  } else if (allDims == 3) {                    \
    RUN_DIR(INDEX_T, 3);                        \
  } else {                                      \
    RUN_DIR(INDEX_T, -1);                       \
  }

#define RUN_T(INDEX_T)                                                  \
  TensorInfo<real, INDEX_T> inputInfo =                                 \
    getTensorInfo<real, THCTensor, INDEX_T>(state, input);              \
  TensorInfo<real, INDEX_T> topKInfo =                                  \
    getTensorInfo<real, THCTensor, INDEX_T>(state, topK);               \
  TensorInfo<int64_t, INDEX_T> indicesInfo =                            \
    getTensorInfo<int64_t, THCudaLongTensor, INDEX_T>(state, indices);  \
                                                                        \
  /* We use these structures solely to find the offset to */            \
  /* each slice we are operating on */                                  \
  inputInfo.sizes[dim] = 1;                                             \
  topKInfo.sizes[dim] = 1;                                              \
  indicesInfo.sizes[dim] = 1;                                           \
                                                                        \
  /* Collapse all other dims */                                         \
  int collapseInputDim = inputInfo.collapseDims(dim);                   \
  int collapseTopKDim = topKInfo.collapseDims(dim);                     \
  int collapseIndicesDim = indicesInfo.collapseDims(dim);               \
                                                                        \
  int64_t inputSlices = 1;                                              \
  for (int i = 0; i < inputInfo.dims; ++i) {                            \
    inputSlices *= inputInfo.sizes[i];                                  \
  }                                                                     \
  int64_t topKSlices = 1;                                               \
  for (int i = 0; i < topKInfo.dims; ++i) {                             \
    topKSlices *= topKInfo.sizes[i];                                    \
  }                                                                     \
                                                                        \
  dim3 grid;                                                            \
  if (!THC_getGridFromTiles(inputSlices, grid)) {                       \
    THError("Slice to sort is too large");                              \
  }                                                                     \
                                                                        \
  dim3 block(std::min(THCRoundUp(sliceSize, (int64_t) 32), (int64_t) 1024)); \
                                                                        \
  /* This is used as a template parameter to calculate indices. */      \
  /* We only specialize it if all collapsed dim sizes are the */        \
  /* same; otherwise, we use -1 which is the specialization */          \
  /* parameter for arbitrary dimensions */                              \
  int allDims = inputInfo.dims;                                         \
  if (topKInfo.dims != allDims || indicesInfo.dims != allDims) {        \
    allDims = -1;                                                       \
  }                                                                     \
                                                                        \
  RUN_DIM(INDEX_T);

  if (THCTensor_nElement(state, input) > 0) {
    // Based on required index size, run the algorithm with the
    // appropriate index type
    if (THCTensor_canUse32BitIndexMath(state, input) &&
        THCTensor_canUse32BitIndexMath(state, topK) &&
        THCTensor_canUse32BitIndexMath(state, indices)) {
      RUN_T(uint32_t);
    } else {
      RUN_T(uint64_t);
    }
  }
#undef RUN_T
#undef RUN_DIM
#undef RUN_DIR
#undef RUN_K

  // Sort the results if the user wants them sorted, since our
  // selection routine does not ensure sorting
  if (sorted) {
    // FIXME: the k/v inplace sort along slice only works for size <=
    // 2048 at the moment
    if (sliceSize <= 2048) {
      // This avoids any memory allocations and performs all sorting
      // work inplace along the slice
      THCTensor_(sortKeyValueInplace)(state, topK, indices, dim, dir);
    } else {
      // Depend upon the backup sort that returns indices, which we
      // can use in conjunction with gather to produce the original
      // indices.
      // This is not the most efficient implementation, especially since
      // there are memory allocations performed here. If the user desires
      // greater performance, they should torch.gather() the results
      // themselves using the reported indices, providing previously
      // allocated tensors to receive the results.
      THCTensor* sortedTopK = THCTensor_(new)(state);
      THCudaLongTensor* sortedIndices = THCudaLongTensor_new(state);
      THCTensor_(sort)(state, sortedTopK, sortedIndices, topK, dim, dir);

      THCudaLongTensor* sortedTopKIndices = THCudaLongTensor_new(state);

      THCudaLongTensor_resizeAs(state, sortedTopKIndices, indices);
      THCudaLongTensor_gather(state, sortedTopKIndices, indices, dim, sortedIndices);

      THCTensor_(freeCopyTo)(state, sortedTopK, topK);
      THCudaLongTensor_freeCopyTo(state, sortedTopKIndices, indices);
      THCudaLongTensor_free(state, sortedIndices);
    }
  }

  THCudaLongTensor_free(state, input);

  THCudaCheck(cudaGetLastError());
}

#endif // THC_GENERIC_FILE
